/*
Copyright (c) 2019 SChernykh

This file is part of RandomX CUDA.

RandomX CUDA is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

RandomX CUDA is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with RandomX CUDA.  If not, see<http://www.gnu.org/licenses/>.
*/


#include "cryptonight.h"
#include "cuda_device.hpp"


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdint>


void randomx_prepare(nvid_ctx *ctx, const void *dataset, size_t dataset_size, uint32_t batch_size)
{
    ctx->rx_batch_size      = batch_size;
    ctx->d_scratchpads_size = batch_size * (ctx->algorithm.l3() + 64);

    if (ctx->rx_dataset_host > 0) {
        CUDA_CHECK(ctx->device_id, hipHostGetDevicePointer(&ctx->d_rx_dataset, const_cast<void *>(dataset), 0));
    }
    else {
        CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_dataset, dataset_size));
        CUDA_CHECK(ctx->device_id, hipMemcpy(ctx->d_rx_dataset, dataset, dataset_size, hipMemcpyHostToDevice));
    }

    CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_long_state, ctx->d_scratchpads_size));
    CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_hashes, batch_size * 64));
    CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_entropy, batch_size * (128 + 2560)));
    CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_vm_states, batch_size * 2560));
    CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_rx_rounding, batch_size * sizeof(uint32_t)));
}
